#include "hip/hip_runtime.h"
/* Copyright 2015 The math21 Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "inner_cc.h"
#include "../operations.h"

namespace math21 {

    // matrix multiplication
    template<typename T>
    __global__ void gpu_matrix_multiply_easy(NumR s, T *a, T *b, T *c, int nr, int r, int nc) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        T sum = 0;
        if (row < nr && col < nc) {
            for (int i = 0; i < r; i++) {
                sum += a[row * r + i] * b[i * nc + col];
            }
            c[row * nc + col] = (T) s * sum;
        }
    }

    // block matrix multiplication
    template<typename T>
    __global__ void gpu_matrix_multiply_shared(NumR s, T *A, T *B, T *C,
                                               int nr_A, int nc_A, int nc_B) {
        __shared__ T sA[MATH21_GPU_BLOCK_SIZE][MATH21_GPU_BLOCK_SIZE];   // Tile size to store elements in shared memory
        __shared__ T sB[MATH21_GPU_BLOCK_SIZE][MATH21_GPU_BLOCK_SIZE];

        int row = blockDim.y * blockIdx.y + threadIdx.y; //To generate ids of threads.
        int col = blockDim.x * blockIdx.x + threadIdx.x;
        T tmp = 0;

        for (int k = 0; k < (((nc_A - 1) / MATH21_GPU_BLOCK_SIZE) + 1); k++) {
            if ((row < nr_A) && (threadIdx.x + (k * MATH21_GPU_BLOCK_SIZE)) < nc_A) {
                sA[threadIdx.y][threadIdx.x] = A[(row * nc_A) + threadIdx.x + (k * MATH21_GPU_BLOCK_SIZE)];
            } else {
                sA[threadIdx.y][threadIdx.x] = 0;
            }
            if (col < nc_B && (threadIdx.y + k * MATH21_GPU_BLOCK_SIZE) < nc_A) {
                sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * MATH21_GPU_BLOCK_SIZE) * nc_B + col];
            } else {
                sB[threadIdx.y][threadIdx.x] = 0;
            }
            __syncthreads();

            for (int j = 0; j < MATH21_GPU_BLOCK_SIZE; ++j) {
                tmp += sA[threadIdx.y][j] * sB[j][threadIdx.x];
            }
            __syncthreads();
        }
        if (row < nr_A && col < nc_B) {
            C[row * nc_B + col] = (T) s * tmp;
        }
    }

    // MATH21_ASSERT(A.isContinuous() && !A.isColumnMajor());
    // MATH21_ASSERT(B.isContinuous() && !B.isColumnMajor());
    template<typename T>
    void _math21_c_matrix_multiply_cuda(NumR s, const Tensor<T> &A, const Tensor<T> &B, Tensor<T> &C) {
        MATH21_ASSERT(!A.isEmpty() && !B.isEmpty(), "empty matrix");
        MATH21_ASSERT(B.nrows() == A.ncols(), "matrix size doesn't match in *");

        NumN nr, nc, r;
        nr = A.nrows();
        nc = B.ncols();
        r = A.ncols();
        if (C.nrows() != nr || C.ncols() != nc) {
            if (nc == 1) {
                C.setSize(nr);
            } else {
                C.setSize(nr, nc);
            }
        }
        MATH21_ASSERT(C.isStandard());

        const T *A_data = math21_memory_tensor_data_address(A);
        const T *B_data = math21_memory_tensor_data_address(B);
        T *C_data = math21_memory_tensor_data_address(C);

        T *d_a, *d_b, *d_c;
        math21_cuda_malloc_device((void **) &d_a, sizeof(T) * nr * r);
        math21_cuda_malloc_device((void **) &d_b, sizeof(T) * r * nc);
        math21_cuda_malloc_device((void **) &d_c, sizeof(T) * nr * nc);

        // copy matrix A and B from host to device memory
        math21_cuda_memcpy_host_to_device(d_a, A_data, sizeof(T) * nr * r);
        math21_cuda_memcpy_host_to_device(d_b, B_data, sizeof(T) * r * nc);

        unsigned int grid_rows = (unsigned int) (nr + MATH21_GPU_BLOCK_SIZE - 1) / MATH21_GPU_BLOCK_SIZE;
        unsigned int grid_cols = (unsigned int) (nc + MATH21_GPU_BLOCK_SIZE - 1) / MATH21_GPU_BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows, 1);
        dim3 dimBlock(MATH21_GPU_BLOCK_SIZE, MATH21_GPU_BLOCK_SIZE, 1);

//        timer t;
//        t.start();

        // Launch kernel
#ifdef MATH21_FLAG_UNDERSTANDABLE
        gpu_matrix_multiply_easy << < dimGrid, dimBlock >> > (s, d_a, d_b, d_c, nr, r, nc);
#else
        gpu_matrix_multiply_shared << < dimGrid, dimBlock >> > (s, d_a, d_b, d_c, nr, r, nc);
#endif

        // Transefr results from device to host
        math21_cuda_memcpy_device_to_host(C_data, d_c, sizeof(T) * nr * nc);
        math21_cuda_DeviceSynchronize();

        math21_cuda_free_device(d_a);
        math21_cuda_free_device(d_b);
        math21_cuda_free_device(d_c);

//        t.end();
//        printf("Time elapsed %f ms.\n\n", t.time());
    }

    namespace detail {
        void _math21_c_matrix_multiply_cuda_Num(NumR s, const TenN &A, const TenN &B, TenN &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }

        void _math21_c_matrix_multiply_cuda_Num(NumR s, const TenZ &A, const TenZ &B, TenZ &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }

        void _math21_c_matrix_multiply_cuda_Num(NumR s, const TenR &A, const TenR &B, TenR &C) {
            _math21_c_matrix_multiply_cuda(s, A, B, C);
        }
    }
}