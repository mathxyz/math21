#include "hip/hip_runtime.h"
/* Copyright 2015 The math21 Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "max_pooling_cuda.h"


__global__ void math21_ml_function_max_pooling_forward_cuda_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad/2;
    int h_offset = -pad/2;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                         cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max_i = (val > max) ? index : max_i;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}

__global__ void math21_ml_function_max_pooling_backward_cuda_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;
    int area = (size-1)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad/2;
    int h_offset = -pad/2;

    float d = 0;
    int l, m;
    for(l = -area; l < area+1; ++l){
        for(m = -area; m < area+1; ++m){
            int out_w = (j-w_offset)/stride + m;
            int out_h = (i-h_offset)/stride + l;
            int out_index = out_w + w*(out_h + h*(k + c*b));
            int valid = (out_w >= 0 && out_w < w &&
                         out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == index) ? delta[out_index] : 0;
        }
    }
    prev_delta[index] += d;
}


void math21_ml_function_max_pooling_forward_cuda(mlfunction_max_pooling *f, const mlfunction_node*finput)
{
    int h = f->out_h;
    int w = f->out_w;
    int c = f->c;

    size_t n = h*w*c*f->batch;

    math21_ml_function_max_pooling_forward_cuda_kernel<<<math21_cuda_gridsize(n), MATH21_CUDA_BLOCK_SIZE >>>(n, f->h, f->w, f->c, f->stride, f->size, f->padding, finput->y, f->output, f->indexes);
    math21_cuda_check_error(hipPeekAtLastError());
}


void math21_ml_function_max_pooling_backward_cuda(mlfunction_max_pooling *f, mlfunction_node*finput)
{
    size_t n = f->h*f->w*f->c*f->batch;

    math21_ml_function_max_pooling_backward_cuda_kernel<<<math21_cuda_gridsize(n), MATH21_CUDA_BLOCK_SIZE >>>(n, f->h, f->w, f->c, f->stride, f->size, f->padding, f->delta, finput->dy, f->indexes);
    math21_cuda_check_error(hipPeekAtLastError());
}