#include "hip/hip_runtime.h"
/* Copyright 2015 The math21 Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <iostream>
#include "inner_cc.h"
#include "cuda/files_c.h"
#include "../matrix/files_c.h"
#include "gpu.h"

namespace math21 {
    void math21_cuda_version() {
        std::cout << "CUDA Compiled version: "
                  << __CUDACC_VER_MAJOR__ << "."
                  << __CUDACC_VER_MINOR__ << "."
                  << __CUDACC_VER_BUILD__
                  << std::endl;

        int runtime_ver;
        hipRuntimeGetVersion(&runtime_ver);
        std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

        int driver_ver;
        hipDriverGetVersion(&driver_ver);
        std::cout << "CUDA Driver version: " << driver_ver << std::endl;

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        int v = prop.major * 10 + prop.minor;
        printf("-gencode arch=compute_%d,code=sm_%d\n", v, v);
    }

    __global__ void _print_hello_world(char *a, int N) {
        char p[12] = "Hello CUDA\n";
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < N) {
            a[idx] = p[idx];
        }
    }

    void math21_cuda_hello_world() {
        char *a_h, *a_d;
        const int N = 12;
        size_t size = N * sizeof(char);
        math21_vector_malloc((void **) &a_h, size);
        math21_cuda_malloc_device((void **) &a_d, size);
        for (int i = 0; i < N; i++) {
            a_h[i] = 0;
        }
        math21_cuda_memcpy_host_to_device(a_d, a_h, size);
        int blocksize = 4;
        int nblock = (N - 1 + blocksize) / blocksize;
        _print_hello_world << < nblock, blocksize >> > (a_d, N);
        math21_cuda_memcpy_device_to_host(a_h, a_d, sizeof(char) * N);
        printf("%s", a_h);
        math21_memory_free(a_h);
        math21_cuda_free_device(a_d);
    }

    void math21_cuda_thrust_version() {
        int major = THRUST_MAJOR_VERSION;
        int minor = THRUST_MINOR_VERSION;

        std::cout << "Thrust v" << major << "." << minor << std::endl;
    }

    __global__ void math21_cuda_atomicAdd_test_kernel(float *foo) {
        atomicAdd(foo, 0.10002);
//        ++(*foo);
    }

    // not tested
    void math21_cuda_atomicAdd_test() {
        float x = 0, *cuda_x;
        hipError_t status = hipMalloc((void **) &cuda_x, sizeof(float));
        math21_cuda_check_error(status);
        hipMemcpy(cuda_x, &x, sizeof(float), hipMemcpyHostToDevice);
        printf("x: %f\n", x);
        math21_cuda_atomicAdd_test_kernel << < 1000, 10 >> > (cuda_x);
        hipMemcpy(&x, cuda_x, sizeof(float), hipMemcpyDeviceToHost);

        char buffer[1024] = {0};
        sprintf(buffer, "0000.bin");
        const char *path = buffer;
        math21_vector_serialize_to_file_wrapper(path, cuda_x, 1);

        hipFree(cuda_x);
        printf("x: %f, sizeof: %ld\n", x, sizeof(x));
    }

}